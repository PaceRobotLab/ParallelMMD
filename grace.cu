#include "hip/hip_runtime.h"

#include "cv2sim\StdAfx.h"
#include "hip/hip_runtime.h"
#include ""
#include <cv.h>
#define TX 32
#define TY 32

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
#if 0
_global_ void kernel(uchar4 *d_out, int w, int h, int2 pos)
{
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	if ((c >= w) || (r >= h)) return;
	const int i = c + r*w;
	const int
}
#endif

void cv2simSmall_grace_cuda(int matches[][4], int nMatches, int height, int width, int step, int channels, double* mWeights,
	char* img1ptr, char* img2ptr, char* maskptr, double eSum, double mSum){

	int offset = 0, newmask, newval;
	double sx = 5.0, sy = 5.0, eGain = 50;
	int j = 0, i = 0, k = 0;
	int id = 0, jd = 0, w = 0;
	for (j = 0; j < height; j++){
		for (i = 0; i < width; i++){
			for (k = 0; k < channels; k++) {
				eSum = 0.0;
				for (w = 0; w < nMatches; w++) {
					id = (i - matches[w][2]); jd = (j - matches[w][3]);
					eSum += (mWeights[w] / mSum)*exp((double)-(id*id / (2.0*sx*sx) + jd*jd / (2.0*sy*sy)));
				}
				eSum *= eGain;

				offset = j*step + i*channels + k;
				newmask = (int)(eSum * 255.0);
				if (newmask > 255) newmask = 255; else if (newmask < 0) newmask = 0;

				if (newmask > 0)
					newval = (int)((double)img1ptr[offset] / (double)maskptr[offset]);
				else newval = img1ptr[offset];

				if (newval > 255) newval = 255; else if (newval < 0) newval = 0;

				img2ptr[offset] = newval; //min( 255, max(1,newval));
				maskptr[offset] = newmask;
				//if (i%2==0) 	fprintf(mlog,"%d,",(unsigned char)maskptr[offset]);
				//if (i==width-1) fprintf(mlog,"0\n");
			}
		}
	}
}

__global__ void kernel_grace(int* matches, int nMatches, int height, int width, int step, int channels, double* mWeights,
	char* img1ptr, char* img2ptr, char* maskptr, double eSum, double mSum){
	

	int offset = 0, newmask, newval;
	double sx = 5.0, sy = 5.0, eGain = 50;
	int j = blockIdx.x, i = threadIdx.x, k = 0;
	int id = 0, jd = 0, w = 0;
	//j = 35, i = 15, k = 0
	//w = 300, h = 500
	//j * w + i = 35 * 300 + 15 = 10515
	eSum = 0.0;
	for (w = 0; w < nMatches; w++) {
		id = (i - matches[w * 4 + 2]); jd = (j - matches[w * 4 + 3]);
		eSum += (mWeights[w] / mSum)*exp((float)-(id*id / (2.0*sx*sx) + jd*jd / (2.0*sy*sy)));
	}
	eSum *= eGain;

	offset = j*step + i*channels + k;
	newmask = (int)(eSum * 255.0);
	if (newmask > 255) newmask = 255; else if (newmask < 0) newmask = 0;

	if (newmask > 0)
		newval = 255-(int)((double)img1ptr[offset] / (double)maskptr[offset]);
	else newval = img1ptr[offset];

	if (newval > 255) newval = 255; else if (newval < 0) newval = 0;

	img2ptr[offset] = newval; //min( 255, max(1,newval));
	maskptr[offset] = newmask;

#if 0
	int ind = threadIdx.x;
	int offset = 0, newmask, newval;
	double sx = 5.0, sy = 5.0, eGain = 50;
	int j = ind, i = 0, k = 0;
	int id = 0, jd = 0, w = 0;
	for (int i = 0; i < width; ++i){
		//j = 35, i = 15, k = 0
		//w = 300, h = 500
		//j * w + i = 35 * 300 + 15 = 10515
		eSum = 0.0;
		for (w = 0; w < nMatches; w++) {
			id = (i - matches[w * 4 + 2]); jd = (j - matches[w * 4 + 3]);
			eSum += (mWeights[w] / mSum)*exp((double)-(id*id / (2.0*sx*sx) + jd*jd / (2.0*sy*sy)));
		}
		eSum *= eGain;

		offset = j*step + i*channels + k;
		newmask = (int)(eSum * 255.0);
		if (newmask > 255) newmask = 255; else if (newmask < 0) newmask = 0;

		if (newmask > 0)
			newval = (int)((double)img1ptr[offset] / (double)maskptr[offset]);
		else newval = img1ptr[offset];

		if (newval > 255) newval = 255; else if (newval < 0) newval = 0;

		img2ptr[offset] = newval; //min( 255, max(1,newval));
		maskptr[offset] = newmask;
	}
#endif
}

void cv2simSmall_grace_cuda_2(int matches[][4], int nMatches, int height, int width, int step, int channels, double* mWeights,
	char* img1ptr, char* img2ptr, char* maskptr, double eSum, double mSum){

	hipError_t cudaStatus;
	char *dev_img1ptr = 0;
	char *dev_img2ptr = 0;
	char *dev_maskptr = 0;
	int *dev_matches = 0;
	double* dev_mWeights = 0;
	GraceTimer timer;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc((void**)&dev_img1ptr, (height*step) * sizeof(char));
	cudaStatus = hipMalloc((void**)&dev_img2ptr, (height*step) * sizeof(char));
	cudaStatus = hipMalloc((void**)&dev_maskptr, (height*step) * sizeof(char));
	cudaStatus = hipMalloc((void**)&dev_matches, nMatches * 4 * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_mWeights, 1000 * sizeof(double));
	//timer.step("malloc fee: %.2f ms\n");

	cudaStatus = hipMemcpy(dev_img1ptr, img1ptr, (height*step) * sizeof(char), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_img2ptr, img2ptr, (height*step) * sizeof(char), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_maskptr, maskptr, (height*step) * sizeof(char), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_matches, matches, nMatches * 4 * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_mWeights, mWeights, 1000 * sizeof(double), hipMemcpyHostToDevice);
	//timer.step("memcpy fee: %.2f ms\n");

	//size_t size = height*width*channels;
	kernel_grace << <height, width >> >(dev_matches, nMatches, height, width, step, channels, dev_mWeights,
		dev_img1ptr, dev_img2ptr, dev_maskptr, eSum, mSum);

	cudaStatus = hipGetLastError();
	cudaStatus = hipDeviceSynchronize();
	//timer.step("call kernel fee: %.2f ms\n");
	cudaStatus = hipMemcpy(maskptr, dev_maskptr, (height*step) * sizeof(char), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(img2ptr, dev_img2ptr, (height*step) * sizeof(char), hipMemcpyDeviceToHost);
	hipFree(dev_img1ptr);
	hipFree(dev_img2ptr);
	hipFree(dev_maskptr);
	hipFree(dev_matches);
	hipFree(dev_mWeights);
	//cudaStatus = hipDeviceReset();
	//timer.step("result fee %.2f ms\n");
}

int calcByCU(){
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 1 + 3;
}

int main11()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
